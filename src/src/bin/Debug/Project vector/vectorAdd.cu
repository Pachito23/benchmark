
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#define N 1000000 //~1GB
#define THREADS 2
#define BLOCKS 2

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i=blockDim.x * blockIdx.x +threadIdx.x;

	while(i<n)
	{
		c[i] = (a[i] + b[i]);
        i+=blockDim.x;
	}
}

int main( void ) {

    clock_t start = clock();
    
    int *a, *b, *c;               // The arrays on the host CPU machine
    int *dev_a, *dev_b, *dev_c;   // The arrays for the GPU device

    // 2.a allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );

    // 2.b. fill the arrays 'a' and 'b' on the CPU with dummy values
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i;
    }

    // 2.c. allocate the memory on the GPU
     hipMalloc( (void**)&dev_a, N * sizeof(int) );
     hipMalloc( (void**)&dev_b, N * sizeof(int) );
     hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // 2.d. copy the arrays 'a' and 'b' to the GPU
     hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice );
     hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice );

    // 3. Execute the vector addition 'kernel function' on th GPU device,
    // declaring how many blocks and how many threads per block to use.
    VectorAdd<<<BLOCKS,THREADS>>>( dev_a, dev_b, dev_c ,N);

    // 4. copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost );

    // verify that the GPU did the work we requested
    bool success = true;
    int total=0;
    printf("Checking %d values in the array.\n", N);
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
        total += 1;
    }
    if (success)  printf( "We did it, %d values correct!\n", total );

    clock_t end = clock();
    int diff = end-start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("Time taken %d seconds %d milliseconds", msec/1000, msec%1000);

    // free the memory we allocated on the CPU
    free( a );
    free( b );
    free( c );

    // free the memory we allocated on the GPU
     hipFree( dev_a );
     hipFree( dev_b );
     hipFree( dev_c );

    return 0;
    }